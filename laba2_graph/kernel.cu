#define _SVID_SOURCE


#include "hip/hip_runtime.h"




#include "omp.h"
#include "stdlib.h"
#include "stdio.h"
#include "time.h"



#pragma warning(disable : 4996)

#define BUFFER_SIZE 255

#define SIZE_CUDA_BLOCK 1024
#define SIZE_CUDA_THREAD_IN_BLOCK 1024


typedef struct Node Node;
typedef struct NodeList NodeList;

struct Node
{
	int value = -1;
	NodeList* paths;

};

struct NodeList
{
	Node* node;
	NodeList* next;
};

struct Graph
{
	int size;
	Node* nodes;
};

struct Pointer
{
	Node* node;
	Pointer* next;
};


int stopwatchWork(int (*onWork)(int), int n) {

	double  startTime = omp_get_wtime();
	int result = (*onWork)(n);
	double endTime = omp_get_wtime();
	float timeWork = (endTime - startTime);
	printf("Time: %f \n", timeWork);

	return result;
}


Graph createGraphFromFile(FILE* file) {
	if (file == NULL)
	{
		puts("File not find");
		exit(1);
	}
	int size;
	Graph graph = Graph();
	fread(&size, sizeof(size), 1, file);
	graph.size = size;
	int sizePaths;
	fread(&sizePaths, sizeof(int), 1, file);
	graph.nodes = new Node[graph.size];
	Node* nodes = graph.nodes;
	for (int i = 0; i < graph.size; i++)
	{
		Node node = Node();
		node.value = i;
		nodes[i] = node;
	}

	for (int row = 0; row < graph.size; row++)
	{
		Node* node = &nodes[row];
		int *hasPaths  = new int[graph.size];
		NodeList* firstNode = NULL;
		NodeList* lastNode = NULL;
		fread(hasPaths, sizeof(int), graph.size, file);
		for (int column = 0; column < graph.size; column++)
		{
			if (hasPaths[column] != 0)
			{
				if (firstNode == NULL)
				{	
					firstNode = new NodeList();
					firstNode->node = &nodes[column];
					node->paths = firstNode;
					lastNode = firstNode;
				}
				else
				{
					NodeList *nextNode = new NodeList();
					nextNode->node = &nodes[column];
					lastNode->next = nextNode;
					lastNode = nextNode;
				}
			}
		}
		delete hasPaths;
	}

	return graph;
}

int main() {
	FILE* file;
	file = fopen("C:\\Users\\webve\\Google Drive\\Universities\\Master Degree\\Chydov\\PP\\laba2_graph\\laba2_graph\\x64\\Debug\\graph_6_5_7.bin", "rb");

	Graph graph= createGraphFromFile(file);
	return(0);
};



