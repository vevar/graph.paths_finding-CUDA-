#include "hip/hip_runtime.h"
#define _SVID_SOURCE


#include "hip/hip_runtime.h"


#include ""

#include "omp.h"
#include "stdlib.h"
#include "stdio.h"
#include "time.h"



#pragma warning(disable : 4996)

#define BUFFER_SIZE 255

#define SIZE_CUDA_BLOCK 1024
#define SIZE_CUDA_THREAD_IN_BLOCK 1024


typedef struct Node Node;
typedef struct NodeList NodeList;

struct Node
{
	int value = -1;
	Node** paths;
	int sizePaths;
};

struct NodeList
{
	Node* node;
	NodeList* next;
};

struct Graph
{
	int size;
	Node* nodes;
};

struct Pointer
{
	Node* node;
	Pointer* next;
};


int stopwatchWork(int (*onWork)(int), int n) {

	double  startTime = omp_get_wtime();
	int result = (*onWork)(n);
	double endTime = omp_get_wtime();
	float timeWork = (endTime - startTime);
	printf("Time: %f \n", timeWork);

	return result;
}

// TODO need refactoring 
Graph createGraphFromFile(FILE* file) {
	if (file == NULL)
	{
		puts("File not find");
		exit(1);
	}
	int size;
	Graph graph = Graph();
	fread(&size, sizeof(size), 1, file);
	graph.size = size;
	int sizePaths;
	fread(&sizePaths, sizeof(int), 1, file);
	graph.nodes = new Node[graph.size];
	Node* nodes = graph.nodes;
	for (int i = 0; i < graph.size; i++)
	{
		Node node = Node();
		node.value = i;
		nodes[i] = node;
	}

	for (int row = 0; row < graph.size; row++)
	{
		Node* node = &nodes[row];
		int *hasPaths  = new int[graph.size];
		NodeList* firstNodeList = NULL;
		NodeList* lastNodeList = NULL;
		int sizeList = 0;
		fread(hasPaths, sizeof(int), graph.size, file);
		for (int column = 0; column < graph.size; column++)
		{
			if (hasPaths[column] != 0)
			{
				if (firstNodeList == NULL)
				{	
					firstNodeList = new NodeList();
					firstNodeList->node = &nodes[column];
					lastNodeList = firstNodeList;
				}
				else
				{
					NodeList *nextNode = new NodeList();
					nextNode->node = &nodes[column];
					lastNodeList->next = nextNode;
					lastNodeList = nextNode;
				}
				sizeList++;
			}
		}
		delete hasPaths;

		node->paths = new Node*[sizeList];
		node->sizePaths = sizeList;
		NodeList *currentNodeList = firstNodeList;
		for (int i = 0; i < node->sizePaths; i++)
		{
			node->paths[i] = currentNodeList->node;
			currentNodeList = currentNodeList->next;
		}
	}

	return graph;
}

int makeStep(Pointer* pointer, Pointer **lastPointer,Node* finishNode, int* pathCounter) {
	int counterNewPointer = 0;
	if (pointer != NULL && pointer->node->sizePaths > 0)
	{
		if (pointer->node == finishNode)
		{
			return counterNewPointer;
		}
		Node* tmpNode = pointer->node;
		Node* nextNode = tmpNode->paths[0];
		if (nextNode == finishNode)
		{
			(*pathCounter)++;
		}
		pointer->node = nextNode;


		for (int indexPath = 1; indexPath < tmpNode->sizePaths; indexPath++)
		{
			nextNode = tmpNode->paths[indexPath];
			if (nextNode == finishNode)
			{
				(*pathCounter)++;
			}
			(*lastPointer)->next = new Pointer();
			lastPointer = &(*lastPointer)->next;
			(*lastPointer)->node = nextNode;
			counterNewPointer++;
		}
	}
	return counterNewPointer;
}

int findAmountPathsGraph(Graph graph, int pointA, int pointB) {
	Node *nodeA = &graph.nodes[pointA];
	Node *nodeB = &graph.nodes[pointB];

	int amountPaths = 0;

	Pointer *movedPointers = new Pointer();
	movedPointers->node = nodeA;
	Pointer* lastPointer = movedPointers;
	int pointerSize = 1;
	for (int step = 0; step < graph.size; step++)
	{
		Pointer* currentPointer = movedPointers;
		int counterNewPointers = 0;
		for (int pointerIndex = 0; pointerIndex < pointerSize; pointerIndex++)
		{
			counterNewPointers +=  makeStep(currentPointer,&lastPointer, nodeB,&amountPaths);
			if (currentPointer == NULL)
			{
				break;
			}
			else
			{
				currentPointer = currentPointer->next;
			}
		}
		pointerSize += counterNewPointers;
	}


	return amountPaths;
}

__global__ void cudaStepAndNewPointers(int startindex, Node* tmpNode, Pointer** lastPointer, Node* finishNode, int* pathCounter, int* counterNewPointer) {
	int indexPath = (blockDim.x * blockIdx.x + threadIdx.x) + startindex;

	Node* nextNode = tmpNode->paths[indexPath];
	if (nextNode == finishNode)
	{
		atomicAdd(pathCounter, 1);
	}
	(*lastPointer)->next = new Pointer();
	lastPointer = &(*lastPointer)->next;
	(*lastPointer)->node = nextNode;
	atomicAdd(counterNewPointer, 1);
	__syncthreads();
}

__global__ void cudaStepForOne(Pointer* pointer, Node* finishNode, int* pathCounter) {
	Node* tmpNode = pointer->node;
	Node* nextNode = tmpNode->paths[0];
	if (nextNode == finishNode)
	{
		atomicAdd(pathCounter, 1);
	}
	pointer->node = nextNode;
	__syncthreads();

}

void cudaMakeStep(Pointer* pointer, Pointer** lastPointer, Node* finishNode, int* pathCounter, int* counterNewPointer) {
	if (pointer != NULL && pointer->node->sizePaths > 0)
	{
		if (pointer->node == finishNode)
		{
			return;
		}
		
		cudaStepForOne<<<1,1>>>(pointer, finishNode, pathCounter);

		Node* tmpNode = pointer->node;
		int sizePaths = tmpNode->sizePaths;
		int amountBlock = sizePaths / SIZE_CUDA_THREAD_IN_BLOCK;
		if ( sizePaths % SIZE_CUDA_THREAD_IN_BLOCK == 0)
		{
			cudaStepAndNewPointers<<<SIZE_CUDA_THREAD_IN_BLOCK, amountBlock>>>(1,tmpNode, lastPointer, finishNode, pathCounter, counterNewPointer);
		}
		else
		{
			if (amountBlock > 1)
			{
				cudaStepAndNewPointers <<<SIZE_CUDA_THREAD_IN_BLOCK, amountBlock >> > (1,tmpNode, lastPointer, finishNode, pathCounter, counterNewPointer);
				cudaStepAndNewPointers <<<sizePaths % SIZE_CUDA_THREAD_IN_BLOCK, 1 >> > (SIZE_CUDA_THREAD_IN_BLOCK * amountBlock + 1, tmpNode, lastPointer, finishNode, pathCounter, counterNewPointer);
			}
			else
			{
				cudaStepAndNewPointers << <sizePaths % SIZE_CUDA_THREAD_IN_BLOCK, 1 >> > (1, tmpNode, lastPointer, finishNode, pathCounter, counterNewPointer);
			}

		}
		
	}
}

int findGPU(Graph graph, int pointA, int pointB) {
	Graph* cudaGraph;
	hipMalloc(&cudaGraph, sizeof(Graph));
	hipMemcpy(cudaGraph, &graph, sizeof(graph), hipMemcpyHostToDevice);




	hipFree(cudaGraph);
}


int cudaFindAmountPathsGraph(Graph graph, int pointA, int pointB) {

	Node* nodeA = &graph.nodes[pointA];
	Node* nodeB = &graph.nodes[pointB];

	int amountPaths = 0;
	int* cudaResult;
	hipMalloc(&cudaResult, sizeof(int));
	hipMemcpy(cudaResult, &amountPaths, sizeof(int), hipMemcpyHostToDevice);

	Pointer* movedPointers = new Pointer();
	movedPointers->node = nodeA;
	Pointer* lastPointer = movedPointers;
	int pointerSize = 1;
	int* cudaCounterNewPointers;
	hipMalloc(&cudaCounterNewPointers, sizeof(int));
	for (int step = 0; step < graph.size; step++)
	{
		Pointer* currentPointer = movedPointers;
		int counterNewPointers = 0;
		

		for (int pointerIndex = 0; pointerIndex < pointerSize; pointerIndex++)
		{
			hipMemcpy(cudaCounterNewPointers, 0, sizeof(int), hipMemcpyHostToDevice);
			cudaMakeStep(currentPointer, &lastPointer, nodeB, cudaResult, cudaCounterNewPointers);
			if (currentPointer == NULL)
			{
				break;
			}
			currentPointer = currentPointer->next;
			int tmpCouter = 0;
			hipMemcpy(&tmpCouter, cudaCounterNewPointers, sizeof(int), hipMemcpyDeviceToHost);
			counterNewPointers += tmpCouter;
		}
		pointerSize += counterNewPointers;
	}

	hipMemcpy(&amountPaths, cudaResult, sizeof(int), hipMemcpyDeviceToHost);

	return amountPaths;
}

int main() {
	FILE* file;
	file = fopen("C:\\University\\Chydov\\PP\\laba2_graph\\laba2_graph\\x64\\Debug\\graph_6_5_7.bin", "rb");

	Graph graph= createGraphFromFile(file);
	
	int pointA;
	int pointB;

	printf("Input point A:");
	scanf("%d", &pointA);

	printf("Input point B:");
	scanf("%d", &pointB);

	int countPathsGPU = cudaFindAmountPathsGraph(graph, pointA, pointB);
	printf("GPU result: %d", countPathsGPU);

	int countPaths = findAmountPathsGraph(graph, pointA, pointB);
	printf("CPU result: %d", countPaths);

	

	return(0);
};



